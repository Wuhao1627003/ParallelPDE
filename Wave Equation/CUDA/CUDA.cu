#include "hip/hip_runtime.h"
#include "..\Sources\Background.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define THREADSPERBLOCK (1 << 2) 
#define NUMBLOCKS (DIST_STEPS / THREADSPERBLOCK)

using namespace std;

__global__ void
init_kernel(float* f, float* g) 
{
    // compute overall index from position of thread in current block,
    // and given the block we are in
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    f[index] = sin(index * XPERIOD);
    g[index] = cos(index * XPERIOD);
}

//modifies g to store second column
__global__ void
secondU_kernel(const float* origU, float* g) 
{
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    g[index] = origU[index] + g[index] * PERIOD;
}

//modifies thisU to store newest value
__global__ void
iterate_kernel(float* thisU, const float* nextU)
{
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    thisU[index] = FAC1 * nextU[index] + FAC2 * ((index > 0 ? nextU[index - 1] : 0) + (index < DIST_STEPS - 1 ? nextU[index + 1] : 0)) - thisU[index];
}

void cudaWrap(float *startU0, float *startU1)
{
    float *prev, *next;
    hipMalloc(&prev, (size_t)(DIST_STEPS) * sizeof(float));
    hipMalloc(&next, (size_t)(DIST_STEPS) * sizeof(float));

    hipMemcpy(prev, startU0, (size_t)DIST_STEPS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(next, startU1, (size_t)DIST_STEPS * sizeof(float), hipMemcpyHostToDevice);

    double startInitTime = CycleTimer::currentSeconds();

    init_kernel<<<NUMBLOCKS, THREADSPERBLOCK>>>(prev, next);
    hipDeviceSynchronize();
    secondU_kernel<<<NUMBLOCKS, THREADSPERBLOCK>>>(prev, next);
    hipDeviceSynchronize();

    double endInitTime = CycleTimer::currentSeconds();
    cout << "Init time: " << (endInitTime - startInitTime) << endl;
    double startIterTime = CycleTimer::currentSeconds();
    double startPartIterTime = startIterTime;
    for (long t = 0; t < MAX_T; t++)
    {
        iterate_kernel<<<NUMBLOCKS, THREADSPERBLOCK>>>(prev, next);
        hipDeviceSynchronize();
        swap(prev, next);
        if (t % (MAX_T >> 3) == 0 && t != 0)
        {
            double endPartIterTime = CycleTimer::currentSeconds();
            cout << "Part Iter time: " << (endPartIterTime - startPartIterTime) << endl;
            startPartIterTime = CycleTimer::currentSeconds();
        }
    }
    
    double endIterTime = CycleTimer::currentSeconds();
    cout << "Total Iter time: " << (endIterTime - startIterTime) << endl;
    hipMemcpy(startU1, next, (size_t)DIST_STEPS * sizeof(float), hipMemcpyDeviceToHost);
    //printArray(startU1, DIST_STEPS);
    hipFree(prev);
    hipFree(next);

}

int main()
{
    float *thisU = (float *)calloc(DIST_STEPS, sizeof(float));
    float *nextU = (float *)calloc(DIST_STEPS, sizeof(float));

    cudaWrap(thisU, nextU);
    
    free(thisU);
    free(nextU);
    return 0;
}