#include "hip/hip_runtime.h"
#include "..\Sources\Background.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADSPERBLOCK (1 << 12) 
#define NUMBLOCKS (DIST_STEPS / THREADSPERBLOCK)

__global__ void
init_kernel(float* f, float* g) 
{
    // compute overall index from position of thread in current block,
    // and given the block we are in
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    f[index] = sin(index * XPERIOD);
    g[index] = cos(index * XPERIOD);
}

//modifies g to store second column
__global__ void
secondU_kernel(const float* origU, float* g) 
{
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    g[index] = origU[index] + g[index] * PERIOD;
}

//modifies thisU to store newest value
__global__ void
iterate_kernel(float* thisU, const float* nextU)
{
    __shared__ float currBlock[THREADSPERBLOCK + 2];

    currBlock[threadIdx.x + 1] = nextU[blockIdx.x * blockDim.x + threadIdx.x];

    if (threadIdx.x == 0)
    {
        if (blockIdx.x != 0)
        {
            currBlock[0] = nextU[blockIdx.x * blockDim.x - 1];
        }
        else
        {
            currBlock[0] = 0;
        }
        if (blockIdx.x != NUMBLOCKS - 1)
        {
            currBlock[blockDim.x + 1] = nextU[(blockIdx.x + 1) * blockDim.x];
        }
        else
        {
            currBlock[blockDim.x + 1] = 0;
        }
    }
    __syncthreads();

    long index = blockIdx.x * blockDim.x + threadIdx.x;
    long i = threadIdx.x;
    thisU[index] = FAC1 * currBlock[i + 1] + FAC2 * (currBlock[i] + currBlock[i + 2]) - thisU[index];
}

void cudaWrap(float *startU0, float *startU1)
{
    float *prev, *next;
    hipMalloc(&prev, (size_t)(DIST_STEPS) * sizeof(float));
    hipMalloc(&next, (size_t)(DIST_STEPS) * sizeof(float));

    hipMemcpy(prev, startU0, (size_t)DIST_STEPS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(next, startU1, (size_t)DIST_STEPS * sizeof(float), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    init_kernel<<<NUMBLOCKS, THREADSPERBLOCK>>>(prev, next);
    hipDeviceSynchronize();
    secondU_kernel<<<NUMBLOCKS, THREADSPERBLOCK>>>(prev, next);
    hipDeviceSynchronize();

    for (long t = 0; t < MAX_T; t++)
    {
        iterate_kernel<<<NUMBLOCKS, THREADSPERBLOCK, (THREADSPERBLOCK + 2) * sizeof(float)>>>(prev, next);
        hipDeviceSynchronize();
        swap(prev, next);
    }
    
    double endTime = CycleTimer::currentSeconds();
    printf("%f\n", endTime - startTime);
    hipMemcpy(startU1, next, (size_t)DIST_STEPS * sizeof(float), hipMemcpyDeviceToHost);
    //printArray(startU1, DIST_STEPS);
    hipFree(prev);
    hipFree(next);
}

int main()
{
    float *thisU = (float *)calloc(DIST_STEPS, sizeof(float));
    float *nextU = (float *)calloc(DIST_STEPS, sizeof(float));

    for (int i = 0; i < NUM_TESTS; i ++)
        cudaWrap(thisU, nextU);
    
    free(thisU);
    free(nextU);
    return 0;
}