#include "hip/hip_runtime.h"
#include "..\Sources\Background.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void
init_kernel(float* f, float* g) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __m128 scalar = _mm_set_ps(XPERIOD, XPERIOD, XPERIOD, XPERIOD);
    __m128 init = _mm_set_ps(i + 3, i + 2, i + 1, i);
    __m128 scaled = _mm_mul_ps(init, scalar);
    __m128 sined = _mm_sin_ps(scaled);
    __m128 cosined = _mm_cos_ps(scaled);
    _mm_stream_ps(f + i, sined);
    _mm_stream_ps(g + i, cosined);
    result[index] = alpha * x[index] + y[index];
}

void init(float* f, float* g)
{
    __m128 scalar = _mm_set_ps(XPERIOD, XPERIOD, XPERIOD, XPERIOD);
    for (long long i = 0; i < DIST_STEPS; i += 4)
    {
        //f[i] = sin(i * XPERIOD);
        //g[i] = cos(i * XPERIOD);
        __m128 init = _mm_set_ps(i + 3, i + 2, i + 1, i);
        __m128 scaled = _mm_mul_ps(init, scalar);
        __m128 sined = _mm_sin_ps(scaled);
        __m128 cosined = _mm_cos_ps(scaled);
        _mm_stream_ps(f + i, sined);
        _mm_stream_ps(g + i, cosined);
    }
}

//modifies g to store second column
void secondU(const float* origU, float* g)
{
    __m128 scalar = _mm_set_ps(PERIOD, PERIOD, PERIOD, PERIOD);
    for (long long i = 0; i < DIST_STEPS / 4; i++)
    {
        __m128 streamU = _mm_load_ps(origU + 4 * i);
        __m128 streamG = _mm_load_ps(g + 4 * i);
        __m128 scaledG = _mm_mul_ps(streamG, scalar);
        __m128 sum = _mm_add_ps(streamU, scaledG);
        _mm_stream_ps(g + 4 * i, sum);
    }
}

//modifies thisU to store newest value
void iterate(float* thisU, float* nextU)
{
    __m128 fac1 = _mm_set_ps(FAC1, FAC1, FAC1, FAC1);
    __m128 fac2 = _mm_set_ps(FAC2, FAC2, FAC2, FAC2);
    
    // float *alwaysThisU = thisU;
    thisU[0] = FAC1 * nextU[0] + FAC2 * nextU[1] - thisU[0];
    thisU[1] = FAC1 * nextU[1] + FAC2 * (nextU[0] + nextU[2]) - thisU[1];
    thisU[2] = FAC1 * nextU[2] + FAC2 * (nextU[1] + nextU[3]) - thisU[2];
    thisU[3] = FAC1 * nextU[3] + FAC2 * (nextU[2] + nextU[4]) - thisU[3];

    float *prev = nextU + 3;
    float *curr = nextU + 4;
    float *next = nextU + 5;
    
    for (long long i = 4; i < DIST_STEPS - 4; i += 4)
    {
        long long j = i - 4;
        __m128 streamNextUNext = _mm_load_ps(next + j);
        __m128 streamNextUCurr = _mm_load_ps(curr + j);
        __m128 streamNextUPrev = _mm_load_ps(prev + j);
        __m128 streamThisU = _mm_load_ps(thisU + i);

        __m128 fac1Prod = _mm_mul_ps(fac1, streamNextUCurr);
        string empty = "";
        __m128 prevNextSum = _mm_add_ps(streamNextUPrev, streamNextUNext);
        __m128 fac2Prod = _mm_mul_ps(fac2, prevNextSum);
        __m128 nextUSum = _mm_add_ps(fac1Prod, fac2Prod);
        __m128 finalThisU = _mm_sub_ps(nextUSum, streamThisU);

        _mm_stream_ps(thisU + i, finalThisU);
        // thisU[i] = FAC1 * nextU[i] + FAC2 * ((i > 0 ? nextU[i - 1] : 0) + (i < DIST_STEPS - 1 ? nextU[i + 1] : 0)) - thisU[i];
    }

    long long endIndex = DIST_STEPS - 4;
    thisU[endIndex] = FAC1 * nextU[endIndex] + FAC2 * (nextU[endIndex - 1] + nextU[endIndex + 1]) - thisU[endIndex];
    thisU[endIndex + 1] = FAC1 * nextU[endIndex + 1] + FAC2 * (nextU[endIndex] + nextU[endIndex + 2]) - thisU[endIndex + 1];
    thisU[endIndex + 2] = FAC1 * nextU[endIndex + 2] + FAC2 * (nextU[endIndex + 1] + nextU[endIndex + 3]) - thisU[endIndex + 2];
    thisU[endIndex + 3] = FAC1 * nextU[endIndex + 3] + FAC2 * nextU[endIndex + 2] - thisU[endIndex + 3];
}

template<class T>
void printArray(T* vec)
{
    for (long long i = 0; i < DIST_STEPS; i ++)
    {
        cout << vec[i] << " ";
    }
    cout << endl;
}

int main()
{
    float *thisU = (float *)malloc(DIST_STEPS * sizeof(float));
    float *nextU = (float *)malloc(DIST_STEPS * sizeof(float));
    double startTime =CycleTimer::currentSeconds();
    init(thisU, nextU);
    secondU(thisU, nextU);
    for (long long t = 0; t < MAX_T; t ++) {
        iterate(thisU, nextU);
        swap(thisU, nextU);
    }
    double endTime =CycleTimer::currentSeconds();
    cout << endTime - startTime << endl;
    //printArray(nextU);
    return 0;
}